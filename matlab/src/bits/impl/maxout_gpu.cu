#include "hip/hip_runtime.h"
// @file maxout_gpu.cu
// @brief Maxout block 
// @author Jia-Ren Chang
// @author Samuel Albanie

/*
Copyright (C) 2017 Jia-Ren Chang and Samuel Albanie.
Licensed under The MIT License [see LICENSE.md for details]
*/

#include "maxout.hpp"
#include <assert.h>
#include <stdio.h>
#include <float.h>
#include <bits/datacu.hpp>
#include <bits/data.hpp>
#include <sm_20_atomic_functions.h>


template<typename T> __global__ void
maxout_kernel(T* pooled,
 const T* data,
 const int pooledWidth,
 const int pooledHeight,
 const int pooledVolume,
 const int numUnits,
 const int numPieces)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (pooledIndex < pooledVolume) {
    /*int px = pooledIndex ;*/
    /*int py = px / pooledWidth ;*/
    /*int pz = py / pooledHeight ;*/
    /*px %= pooledWidth ;*/
    /*py %= pooledHeight ;*/
    /*data += pz * (pooledWidth * pooledHeight) ;*/

    printf("pooledIndex: %d\n", pooledIndex) ;
    /*printf("pooledVolume: %d\n", pooledVolume) ;*/
    int area = pooledWidth * pooledHeight ;
    int s = pooledIndex % area ;  // spatial offset
    int u  = (pooledIndex / area) % numUnits ; // unit
    int t = pooledIndex / (area * numUnits) ; // trial 
    int offset = area * (u + t * numUnits * numPieces) ; // channel offset
    printf("s: %d\n", s) ;
    printf("u: %d\n", u) ;
    printf("t: %d\n", t) ;
    printf("offset: %d\n", offset) ;
    printf("numPieces: %d\n", numPieces) ;
    T bestValue = data[offset + s] ;  
    for (int k = 0; k < numPieces ; ++k) {     
       int idx = area*(u + k*numUnits + t*numUnits*numPieces) + s ;
       bestValue = max(bestValue, data[idx]) ;
       printf("k: %d, idx: %d, p: %d, best: %g\n", k, idx, pooledIndex, bestValue) ;
    }
    pooled[pooledIndex] = bestValue ;
    printf("storing: %g at pooledIndex: %d \n", pooled[pooledIndex], pooledIndex) ;
  }
}

template<typename T> __global__ void
maxout_backward_kernel(T* derData,
 const T* data,
 const T* derPooled,
 const int pooledWidth,
 const int pooledHeight,
 const int pooledVolume,
 const int numUnits,
 const int numPieces)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  printf("pooledIndex: %d\n", pooledIndex) ;
  if (pooledIndex < pooledVolume) {
    int thx = pooledIndex % (pooledWidth*pooledHeight);  // which element in pooled
    int ut  = (pooledIndex / (pooledWidth*pooledHeight)) % numUnits; //which unit
    int ntr = pooledIndex / (pooledWidth*pooledHeight*numUnits); // which trial   
    T bestValue = data[thx + pooledWidth*pooledHeight*(ut +  ntr*numUnits*numPieces)];  // GET value in data
    int bestindex = 0;    
    for (int k = 0; k < numPieces ; ++k) {
      //T value = data[thx + pooledWidth*pooledHeight*(ut*numPieces+k)];
      T value = data[thx + pooledWidth*pooledHeight*(ut + k*numUnits +  ntr*numUnits*numPieces)];
      if (value > bestValue) {
        bestValue = value ;
        bestindex = k;    
      }
    }
    /*
     Comment (from original pooling implementation): 
     This is bad, but required to eliminate a race condition when writing
     to bottom_diff.
     Caffe goes the other way around, but requrires remembering the layer
     output, or the maximal indexes.
     atomicAdd(add, val)
     */
    int dain = thx + pooledWidth*pooledHeight*(ut + bestindex*numUnits +  ntr*numUnits*numPieces);
    atomicAdd(derData + dain, derPooled[pooledIndex]) ;
    //derData[dain] = derPooled[pooledIndex];
  }
}

/* ---------------------------------------------------------------- */
/*                                                   maxout_forward */
/* ---------------------------------------------------------------- */

namespace vl { namespace impl {

    template<typename T>
    struct maxout<vl::VLDT_GPU,T>
    {

    static vl::ErrorCode
    forward(float* pooled,
            float const* data,
            size_t height, size_t width, size_t depth,
            size_t numUnits, size_t numPieces)
    {
      int pooledWidth = width;
      int pooledHeight = height;
      int pooledVolume = pooledWidth * pooledHeight * depth / numPieces ;
      maxout_kernel<float>
        <<< vl::divideAndRoundUp(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
        (pooled, data, pooledHeight, pooledWidth, pooledVolume, numUnits, numPieces) ;
      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }

/* ---------------------------------------------------------------- */
/*                                                  maxout_backward */
/* ---------------------------------------------------------------- */ static vl::ErrorCode
    backward(float* derData,
             float const* data,
             float const* derPooled,
             size_t height, size_t width, 
             size_t depth, size_t numUnits, 
             size_t numPieces)
    {
      int pooledWidth = width;
      int pooledHeight = height;
      int pooledVolume = pooledWidth * pooledHeight * depth /  numPieces;
      maxout_backward_kernel<float>
      <<< vl::divideAndRoundUp(pooledVolume, VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>
          (derData, data, derPooled,
           pooledHeight, pooledWidth, pooledVolume,
            numUnits, numPieces);

      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }
} ;

} } // namespace vl::impl

template struct vl::impl::maxout<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::maxout<vl::VLDT_GPU, double> ;
#endif
