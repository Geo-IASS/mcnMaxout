#include "hip/hip_runtime.h"
// @file maxout_gpu.cu
// @brief Maxout block 
// @author Jia-Ren Chang
// @author Samuel Albanie

/*
Copyright (C) 2017 Jia-Ren Chang and Samuel Albanie.
Licensed under The MIT License [see LICENSE.md for details]
*/

#include "maxout.hpp"
#include <assert.h>
#include <stdio.h>
#include <float.h>
#include <bits/datacu.hpp>
#include <bits/data.hpp>
#include <sm_20_atomic_functions.h>


template<typename T> __global__ void
maxout_kernel(T* pooled,
              const T* data,
              const int pooledWidth,
              const int pooledHeight,
              const int pooledVolume,
              const int numUnits,
              const int numPieces)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (pooledIndex < pooledVolume) {
    int area = pooledWidth * pooledHeight ;
    int s = pooledIndex % area ;  // spatial offset
    int u  = (pooledIndex / area) % numUnits ; // unit index
    int b = pooledIndex / (area * numUnits) ; // batch index 
    int offset = area * (u * numPieces + b * numUnits * numPieces) ; 
    T bestValue = data[offset + s] ;  
    for (int k = 0; k < numPieces ; ++k) {     
       int idx = area*(k + u*numPieces + b*numUnits*numPieces) + s ;
       bestValue = max(bestValue, data[idx]) ;
    }
    pooled[pooledIndex] = bestValue ;
  }
}

template<typename T> __global__ void
maxout_backward_kernel(T* derData,
                      const T* data,
                      const T* derPooled,
                      const int pooledWidth,
                      const int pooledHeight,
                      const int pooledVolume,
                      const int numUnits,
                      const int numPieces)
{
  int pooledIndex = threadIdx.x + blockIdx.x * blockDim.x;
  if (pooledIndex < pooledVolume) {
    int area = pooledWidth * pooledHeight ;
    int s = pooledIndex % area ;  // spatial offset
    int u  = (pooledIndex / area) % numUnits ; // unit index
    int b = pooledIndex / (area * numUnits) ; // batch index 
    int offset = area * (u * numPieces + b * numUnits * numPieces) ; 
    T bestValue = data[offset + s] ;  
    int bestIndex = 0;    
    for (int k = 0; k < numPieces ; ++k) {
      int idx = area*(k + u*numPieces + b*numUnits*numPieces) + s ;
      T value = data[idx];
      if (value > bestValue) {
        bestValue = value ;
        bestIndex = k;    
      }
    }
    /*
     Comment (from original pooling implementation): 
     This is bad, but required to eliminate a race condition when writing
     to bottom_diff.
     Caffe goes the other way around, but requrires remembering the layer
     output, or the maximal indexes.
     atomicAdd(add, val)
     */
    int dain = offset + s + area * bestIndex ;
    atomicAdd(derData + dain, derPooled[pooledIndex]) ;
  }
}

/* ---------------------------------------------------------------- */
/*                                                   maxout_forward */
/* ---------------------------------------------------------------- */

namespace vl { namespace impl {

  template<typename T>
  struct maxout<vl::VLDT_GPU,T>
  {
    static vl::ErrorCode
    forward(T* pooled,
            T const* data,
            size_t height, size_t width, size_t depth,
            size_t numUnits, size_t numPieces)
    {
      int pooledWidth = width;
      int pooledHeight = height;
      int pooledVolume = pooledWidth * pooledHeight * depth / numPieces ;
      maxout_kernel<T><<< vl::divideAndRoundUp(pooledVolume, 
        VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>(pooled, data, 
          pooledHeight, pooledWidth, pooledVolume, numUnits, numPieces) ;
      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }

/* ---------------------------------------------------------------- */
/*                                                  maxout_backward */
/* ---------------------------------------------------------------- */ 
    static vl::ErrorCode
    backward(T* derData,
             T const* data,
             T const* derPooled,
             size_t height, size_t width, 
             size_t depth, size_t numUnits, 
             size_t numPieces)
    {
      int pooledWidth = width;
      int pooledHeight = height;
      int pooledVolume = pooledWidth * pooledHeight * depth /  numPieces;
      maxout_backward_kernel<T><<< vl::divideAndRoundUp(pooledVolume, 
        VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS >>>(derData, data, 
        derPooled, pooledHeight, pooledWidth, pooledVolume, numUnits, 
          numPieces) ;
      hipError_t status = hipPeekAtLastError() ;
      return (status == hipSuccess) ? vl::VLE_Success : vl::VLE_Cuda ;
    }
  } ;

} } // namespace vl::impl

template struct vl::impl::maxout<vl::VLDT_GPU, float> ;

#ifdef ENABLE_DOUBLE
template struct vl::impl::maxout<vl::VLDT_GPU, double> ;
#endif
